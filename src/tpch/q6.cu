// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <string>
#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "tpch_utils.h"

using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;

/**
 * Globals, constants and typedefs
 */
bool                    g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

float runQuery(int* d_l_shipdate, int* d_l_discount, int* d_l_quantity, int lo_num_entries, 
    hipcub::CachingDeviceAllocator&  g_allocator) {

    // Setup all the timing stuff
    hipEvent_t gpu_start, gpu_stop; 
    hipEventCreate(&gpu_start); 
    hipEventCreate(&gpu_stop);
    high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_ms;
    start = high_resolution_clock::now();
    hipEventRecord(gpu_start, 0);

    // Allocate space for the result on the GPU
    unsigned long long* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_sum, sizeof(long long)));
    hipMemset(d_sum, 0, sizeof(long long));

    // Record that the kernel has finished
    float time_query;
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&time_query, gpu_start, gpu_stop);

    // Get the total amount of reveue
    unsigned long long revenue;
    CubDebugExit(hipMemcpy(&revenue, d_sum, sizeof(long long), hipMemcpyDeviceToHost));

    // Record that we finished the query
    end = high_resolution_clock::now();
    duration_ms = std::chrono::duration_cast<duration<double, std::milli>>(end - start);

    // Log the results
    std::cout << "Revenue: " << revenue << std::endl;
    std::cout << "Time taken GPU: " << time_query << " ms" << std::endl;
    std::cout << "Time taken total: " << duration_ms.count() << " ms" << std::endl;

    CLEANUP(d_sum);
    return time_query;
}
/**
 * Main
 */
int main(int argc, char** argv)
{
  // Get the data directory
  CommandLineArgs args(argc, argv);
  string data_dir = std::string(BASE_PATH) + std::string(DATA_DIR);
  
  // Read the columns
  int *h_l_shipdate = loadColumn<int>(data_dir, "lineitem", "l_shipdate", LO_LEN);
  int *h_l_discount = loadColumn<int>(data_dir, "lineitem", "l_discount", LO_LEN);
  int *h_l_quantity = loadColumn<int>(data_dir, "lineitem", "l_quantity", LO_LEN);
  std::cout << "** Loaded all the data **" << std::endl;

  // Begin the debugger
  CubDebugExit(args.DeviceInit());

  // Move the data to the GPU
  int *d_l_shipdate = loadToGPU<int>(h_l_shipdate, LO_LEN, g_allocator);
  int *d_l_discount = loadToGPU<int>(h_l_discount, LO_LEN, g_allocator);
  int *d_l_quantity = loadToGPU<int>(h_l_quantity, LO_LEN, g_allocator);
  std::cout << "** Loaded all the data to the GPU **" << std::endl;

  // Run the query
  runQuery(d_l_shipdate, d_l_discount, d_l_quantity, LO_LEN, g_allocator);

  return 0;
}

