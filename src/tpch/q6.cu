#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <string>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "tpch_utils.h"

using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;

#define NUM_BLOCKS 640
#define NUM_TRIALS 10

/**
 * Globals, constants and typedefs
 */
bool  g_verbose = true;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

__global__ void QueryKernel(int* d_l_shipdate, float* d_l_discount, float* d_l_quantity, 
float* d_l_extendedprice, float* total, int lo_num_entries) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    while(idx < lo_num_entries) {
      // Check the conditions for the query
      int curr_ship_date = d_l_shipdate[idx]; 
      if(curr_ship_date >= 8766 && curr_ship_date < 9131) {
        float curr_discount = d_l_discount[idx];
        if(curr_discount >= 0.05 && curr_discount <= 0.070001) {
          if(d_l_quantity[idx] < 24.0) {
            atomicAdd(total, d_l_extendedprice[idx] * curr_discount);
          }
        }
      }
      
      idx += gridDim.x * blockDim.x;
    }
}

float runQuery(int* d_l_shipdate, float* d_l_discount, float* d_l_quantity, float* d_l_extendedprice, 
int lo_num_entries, hipcub::CachingDeviceAllocator&  g_allocator, float* results, float* timings, int trial_num) {
    // Setup all the timing stuff
    hipEvent_t gpu_start, gpu_stop; 
    hipEventCreate(&gpu_start); 
    hipEventCreate(&gpu_stop);
    high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_ms;
    start = high_resolution_clock::now();
    hipEventRecord(gpu_start, 0);

    // Allocate space for the result on the GPU
    float* total = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**) &total, sizeof(float)));
    hipMemset(total, 0, sizeof(float));

    // Run the kernel
    QueryKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_l_shipdate, d_l_discount, d_l_quantity, 
    d_l_extendedprice, total, LO_LEN);

    // Record that the kernel has finished
    float time_query;
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&time_query, gpu_start, gpu_stop);

    // Get the total amount of reveue
    float revenue;
    CubDebugExit(hipMemcpy(&revenue, total, sizeof(float), hipMemcpyDeviceToHost));

    // Record that we finished the query
    end = high_resolution_clock::now();
    duration_ms = std::chrono::duration_cast<duration<double, std::milli>>(end - start);

    // Log the results
    results[trial_num] = revenue;
    timings[2 * trial_num] = time_query;
    timings[2 * trial_num + 1] = (float) duration_ms.count();

    CubDebugExit(g_allocator.DeviceFree(total));
    return time_query;
}

#define MAX_ALLOWED_DIFF 500
void verifySameResult(float* results, int num_trials) {
  float expected = results[0];
  for(int i = 1; i < num_trials; i++) {
    float diff = std::abs(expected - results[i]);
    if(diff > MAX_ALLOWED_DIFF) {
      std::cerr << "Idx " << i << " has value " << results[i] << " but idx 0 has value " << expected 
      << ", which have a diff of " << diff << std::endl;
      exit(1);
    }
  }
}

void printTimings(float* timings, int num_trials) {
  float gpuTotalTime = 0.0;
  float overallTotalTime = 0.0;
  for(int i = 0; i < num_trials; i += 2) {
    gpuTotalTime += timings[i];
    overallTotalTime += timings[i + 1];
  }
  std::cout << "Average time taken GPU: " << gpuTotalTime/num_trials << std::endl;
  std::cout << "Average time taken overall: " << overallTotalTime/num_trials << std::endl;
} 

/**
 * Main
 */
int main(int argc, char** argv)
{
  // Get the data directory
  CommandLineArgs args(argc, argv);
  string data_dir = std::string(BASE_PATH) + std::string(DATA_DIR);
  
  // Read the columns
  int *h_l_shipdate = loadColumn<int>(data_dir, "lineitem", "l_shipdate", LO_LEN);
  float *h_l_discount = loadColumn<float>(data_dir, "lineitem", "l_discount", LO_LEN);
  float *h_l_quantity = loadColumn<float>(data_dir, "lineitem", "l_quantity", LO_LEN);
  float *h_l_extendedprice = loadColumn<float>(data_dir, "lineitem", "l_extendedprice", LO_LEN);

  // Begin the debugger
  CubDebugExit(args.DeviceInit());

  // Copy the data to the GPU
  int *d_l_shipdate = loadToGPU<int>(h_l_shipdate, LO_LEN, g_allocator);
  float *d_l_discount = loadToGPU<float>(h_l_discount, LO_LEN, g_allocator);
  float *d_l_quantity = loadToGPU<float>(h_l_quantity, LO_LEN, g_allocator);
  float *d_l_extendedprice = loadToGPU<float>(h_l_extendedprice, LO_LEN, g_allocator);

  // Perform multiple trials
  float* results = new float[NUM_TRIALS];
  float* timings = new float[2 * NUM_TRIALS];
  for(int trial_num = 0; trial_num < NUM_TRIALS; trial_num++) {
    runQuery(d_l_shipdate, d_l_discount, d_l_quantity, d_l_extendedprice, 
    LO_LEN, g_allocator, results, timings, trial_num);
  }

  // Output the timings
  verifySameResult(results, NUM_TRIALS);
  printTimings(timings, NUM_TRIALS);

  // Free the memory
  CubDebugExit(hipFree(d_l_shipdate));
  CubDebugExit(hipFree(d_l_discount)); 
  CubDebugExit(hipFree(d_l_quantity));
  CubDebugExit(hipFree(d_l_extendedprice));  
  return 0;
}

